#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>
#include <vector>
#include "Constants.h"
#include "particle.h"
#include "particleSystem.h"

#define cudaCheck(stmt) do {													\
	hipError_t err = stmt;														\
	if (err != hipSuccess) {													\
		fprintf(stderr, "Failed to run stmt ", #stmt);							\
		fprintf(stderr, "Got CUDA error ... %s\n", hipGetErrorString(err));	\
	}																			\
} while (0);

#define BLOCK_SIZE 256

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void gravitySerial(std::vector<particle> particles);
void gravityWithCuda(particle* particles, int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void gravKernel(particle* particles, int size, int simul_length) {
	
	//todo: kernel where each thread handles/updated a single particle
	//hard part: make this work across blocks like in mp 5.2 (not sure if this is doable)

	//__shared__ float particles_shared[BLOCK_SIZE];
	//above DOES NOT COMPILE (shared array of objects)
	//see Dynamic Shared Memory: http://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
	//see response: http://stackoverflow.com/questions/27230621/cuda-shared-memory-inconsistent-results

	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < size) {
		//load phase
		//particles_shared[threadIdx.x] = particles[i];
		__syncthreads();

	}
}

int main()
{

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaCheck(hipDeviceReset());

	particleSystem parSys(NUM_PARTICLES);
	parSys.printParticles();
	double* pos = parSys.particlesPosDoubleArray();
	double* vel = parSys.particlesVelDoubleArray();
	parSys.printPosDoubleArray(pos);
	parSys.printVelDoubleArray(vel);
	//parSys.gravitySerial(SIMULATION_LENGTH);


	system("pause"); //see output of terminal
	return 0;
}

void gravitySerial(std::vector<particle> particles) {
	int counter = 0;
	while (counter < SIMULATION_LENGTH) {
		for (std::vector<particle>::iterator it = particles.begin(); it != particles.end(); ++it) {
			v3 force = v3(0.0, 0.0, 0.0);
			for (std::vector<particle>::iterator itt = particles.begin(); itt != particles.end(); ++itt) {
				if (it != itt) {
					// force on i (it) by j (itt)
					v3 currRay = it->getRay(*itt);
					double dist = it->getDistance(*itt);
					double mi = it->getMass();
					double mj = itt->getMass();
					force.x += (double)GRAVITY * (double)mj * (double)currRay.x / (double)pow(dist, 3.0);
					force.y += (double)GRAVITY * (double)mj * (double)currRay.y / (double)pow(dist, 3.0);
					force.z += (double)GRAVITY * (double)mj * (double)currRay.z / (double)pow(dist, 3.0);
				}
			}
			it->updateParticle(EPOCH, force);
			it->printProps();
			std::cout << "Distance: " << it->getDistance(*(it++)) << std::endl;
		}
		counter++;
	}
}

void gravityWithCuda(particle *particles, int size) {
	particle *particles_device;

	cudaCheck(hipSetDevice(0)); //choose which GPU to run on
	cudaCheck(hipMalloc((void **)&particles_device, size * sizeof(particle)));
	cudaCheck(hipMemcpy(particles_device, particles, size * sizeof(particle), hipMemcpyHostToDevice));

	dim3 dimGrid;
	dim3 dimBlock;
	dimGrid.x = (size - 1) / BLOCK_SIZE + 1;
	dimBlock.x = BLOCK_SIZE;

	gravKernel<<<dimGrid,dimBlock,dimBlock.x*sizeof(particle)>>>(particles_device, size, SIMULATION_LENGTH);

	cudaCheck(hipDeviceSynchronize());
	cudaCheck(hipMemcpy(particles, particles_device, size * sizeof(particle), hipMemcpyDeviceToHost));
	cudaCheck(hipFree(particles_device));
	return;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}