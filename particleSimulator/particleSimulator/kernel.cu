#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <device_atomic_functions.h>
#include <>

#include <iostream>
#include <stdio.h>
#include <vector>
#include "Constants.h"
#include "particle.h"
#include "particleSystem.h"

#define cudaCheck(stmt) do {													\
	hipError_t err = stmt;														\
	if (err != hipSuccess) {													\
		fprintf(stderr, "Failed to run stmt ", #stmt); 							\
		fprintf(stderr, "Got CUDA error ... %s\n", hipGetErrorString(err)); 	\
	}																			\
} while (0);

__constant__ float GRAVITY_CUDA = 100.066742f; //KEEP THIS THE SAME AS ITS CONSTANTS_H COUNTERPART!!!

//calculate forces and resultant acceleration for a SINGLE particle due to physics interactions with ALL particles in system
//also updates positions and velocities
__global__
void gravityParallelKernel(float* positions, float* velocities, float* accelerations, unsigned int simulationLength) {

	//strategy: one thread (id) per particle

	__shared__ float3 particles_shared[BLOCK_SIZE];
	__shared__ float3 velocities_shared[BLOCK_SIZE];
	__shared__ float3 accelerations_shared[BLOCK_SIZE];

	__shared__ float3 particles_temp[BLOCK_SIZE];
	__shared__ float3 velocities_temp[BLOCK_SIZE];
	__shared__ float3 accelerations_temp[BLOCK_SIZE];

	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= NUM_PARTICLES) return;

	//LOAD PHASE via float3 conversion
	float3 pos, vel, acc;
	pos.x = positions[3 * id];
	pos.y = positions[3 * id + 1];
	pos.z = positions[3 * id + 2];
	particles_shared[threadIdx.x] = pos;
	vel.x = velocities[3 * id];
	vel.y = velocities[3 * id + 1];
	vel.z = velocities[3 * id + 2];
	velocities_shared[threadIdx.x] = vel;
	acc.x = accelerations[3 * id];
	acc.y = accelerations[3 * id + 1];
	acc.z = accelerations[3 * id + 2];
	accelerations_shared[threadIdx.x] = acc;
	
	if (PARALLEL_DEBUG) {
		printf("import - id: %d\tpos: (%f, %f, %f)\tvel: (%f, %f, %f)\tacc:(%f, %f, %f)\n", id, pos.x, pos.y, pos.z, vel.x, vel.y, vel.z, acc.x, acc.y, acc.z);
	}
	__syncthreads();

	//CALCULATION PHASE
	for (unsigned int simCount = 0; simCount < simulationLength; simCount++) {
		//acc calculation phase
		for (unsigned int i = 0; i < BLOCK_SIZE; i++) {
			particles_temp[i] = particles_shared[i];
			velocities_temp[i] = velocities_shared[i];
			accelerations_temp[i] = accelerations_shared[i];
		}
		__syncthreads();
		float3 curr = particles_temp[id]; //current position for given iteration
		float3 force = { 0.0f, 0.0f, 0.0 };
		for (unsigned i = 0; i < BLOCK_SIZE && i < NUM_PARTICLES; i++) { //all (other) particles
			//float3 other = particles_shared[i];
			float3 other = particles_temp[i];
			if (id != i) /*(curr.x != other.x || curr.y != other.y || curr.z != other.z)*/ { //don't affect own particle
				float3 ray = { curr.x - other.x, curr.y - other.y, curr.z - other.z };
				if (PARALLEL_DEBUG) {
					printf("ray (%u,%u); (%f,%f,%f)\n", id, i, ray.x, ray.y, ray.z);
				}
				float dist = (curr.x - other.x)*(curr.x - other.x) + (curr.y - other.y)*(curr.y - other.y) + (curr.z - other.z)*(curr.z - other.z);
				dist = sqrt(dist);
				if (PARALLEL_DEBUG) {
					printf("distance (%u,%u); %f\n", id, i, dist);
				}
				float xadd = GRAVITY_CUDA * UNIVERSAL_MASS * (float)ray.x / (dist * dist * dist);
				float yadd = GRAVITY_CUDA * UNIVERSAL_MASS * (float)ray.y / (dist * dist * dist);
				float zadd = GRAVITY_CUDA * UNIVERSAL_MASS * (float)ray.z / (dist * dist * dist);
				if (PARALLEL_DEBUG) {
					printf("(xadd, yadd, zadd) (%u,%u); (%f,%f,%f)\n", id, i, xadd, yadd, zadd);
				}

				force.x += xadd / UNIVERSAL_MASS;
				force.y += yadd / UNIVERSAL_MASS;
				force.z += zadd / UNIVERSAL_MASS;

				__syncthreads(); //this shit's important

				//update phase
				particles_shared[id].x += velocities_temp[id].x * EPOCH; //EPOCH is dt
				particles_shared[id].y += velocities_temp[id].y * EPOCH;
				particles_shared[id].z += velocities_temp[id].z * EPOCH;
				curr = particles_shared[id]; //for next iteration (update current position)

				velocities_shared[id].x += accelerations_temp[id].x * EPOCH; //EPOCH is dt
				velocities_shared[id].y += accelerations_temp[id].y * EPOCH;
				velocities_shared[id].z += accelerations_temp[id].z * EPOCH;

				//this is why that shit's important
				accelerations_shared[id].x = force.x; //EPOCH is dt
				accelerations_shared[id].y = force.y;
				accelerations_shared[id].z = force.z;

			}
		}
		if (PARALLEL_UPDATE_OUTPUT) {
			printf("update (%d)\tpos: (%f, %f, %f)\tvel: (%f, %f, %f)\tacc:(%f, %f, %f)\n", id, particles_shared[id].x, particles_shared[id].y, particles_shared[id].z,
				velocities_shared[id].x, velocities_shared[id].y, velocities_shared[id].z,
				accelerations_shared[id].x, accelerations_shared[id].y, accelerations_shared[id].z);
		}
			
		__syncthreads();
	}

	//OUTPUT PHASE via float conversion
	positions[3 * id] = particles_shared[id].x;
	positions[3 * id + 1] = particles_shared[id].y;
	positions[3 * id + 2] = particles_shared[id].z;
	velocities[3 * id] = velocities_shared[id].x;
	velocities[3 * id + 1] = velocities_shared[id].y;
	velocities[3 * id + 2] = velocities_shared[id].z;
	accelerations[3 * id] = accelerations_shared[id].x;
	accelerations[3 * id + 1] = accelerations_shared[id].y;
	accelerations[3 * id + 2] = accelerations_shared[id].z;
}

void gravityParallel(float* hostPositions, float* hostVelocities, float* hostAccelerations, unsigned int simulationLength) {
	//CUDA prep code
	float* devicePositions;
	float* deviceVelocities;
	float* deviceAccelerations;
	size_t size = NUM_PARTICLES * 3 * sizeof(float);

	cudaCheck(hipSetDevice(0)); //choose GPU
	cudaCheck(hipMalloc((void **)&devicePositions, size));
	cudaCheck(hipMalloc((void **)&deviceVelocities, size));
	cudaCheck(hipMalloc((void **)&deviceAccelerations, size));
	cudaCheck(hipMemcpy(devicePositions, hostPositions, size, hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(deviceVelocities, hostVelocities, size, hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(deviceAccelerations, hostAccelerations, size, hipMemcpyHostToDevice));
	dim3 dimGrid, dimBlock;
	dimGrid.x = (size - 1) / BLOCK_SIZE + 1;
	dimBlock.x = BLOCK_SIZE;
	gravityParallelKernel <<<dimGrid, dimBlock >>>(devicePositions, deviceVelocities, deviceAccelerations, simulationLength);
	cudaCheck(hipDeviceSynchronize());
	cudaCheck(hipMemcpy(hostPositions, devicePositions, size, hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(hostVelocities, deviceVelocities, size, hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(hostAccelerations, deviceAccelerations, size, hipMemcpyDeviceToHost));
	cudaCheck(hipFree(devicePositions));
	cudaCheck(hipFree(deviceVelocities));
	cudaCheck(hipFree(deviceAccelerations));
	
	return;
}

//print particles after a single round of serial and parallel to compare output and check correctness
void particleSystem::gravityBoth(float* positions, float* velocities, float* accelerations, unsigned int numRounds) {
	unsigned int round;
	for (round = 0; round < numRounds; round++) {
		
		//SERIAL PORTION
		std::cout << "Serial round " << round << std::endl;
		this->gravitySerial(1); //execution phase
		//this->printParticles(); //print phase
		std::cout << std::endl;

		//PARALLEL PORTION
		std::cout << "Parallel round " << round << std::endl;
		gravityParallel(positions, velocities, accelerations, 1); //execution phase
		//printParticlcesArrays(positions, velocities, accelerations); //print phase
		std::cout << std::endl;
	}

	//CUDA cleanup code
}

int main()
{

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaCheck(hipDeviceReset());
	std::cout << "Initizing Particle System..." << std::endl;
	particleSystem parSys(NUM_PARTICLES);
	parSys.printParticles();
	//parSys.gravitySerial(SIMULATION_LENGTH);
	float* pos = parSys.particlesPosfloatArray();
	float* vel = parSys.particlesVelfloatArray();
	float* acc = parSys.particlesAccfloatArray();
	std::cout << std::endl;
	//parSys.printPosFloatArray(pos);
	//parSys.printVelFloatArray(vel);
	//parSys.printAccFloatArray(acc);
	parSys.gravityBoth(pos, vel, acc, SIMULATION_LENGTH);

	//parSys.gravitySerial(SIMULATION_LENGTH);
	//gravityParallel(pos, vel, acc, SIMULATION_LENGTH);

	system("pause"); //see output of terminal
	return 0;
}
