#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <device_atomic_functions.h>
#include <>

#include <iostream>
#include <stdio.h>
#include <vector>
#include "Constants.h"
#include "particle.h"
#include "particleSystem.h"

#define cudaCheck(stmt) do {													\
	hipError_t err = stmt;														\
	if (err != hipSuccess) {													\
		fprintf(stderr, "Failed to run stmt ", #stmt);							\
		fprintf(stderr, "Got CUDA error ... %s\n", hipGetErrorString(err));	\
	}																			\
} while (0);

//calculate forces and resultant acceleration for a SINGLE particle due to physics interactions with ALL particles in system
__device__
void gravityParallelAccelKernel(float3 curr, float* positions, unsigned int simulationLength, float3 &accel) {

	//strategy: one thread per particle

	__shared__ float3 particles_shared[BLOCK_SIZE];
	accel = { 0.0f, 0.0f, 0.0 };

	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= NUM_PARTICLES) return;

	//load phase
	float3 pos;
	pos.x = positions[3 * id];
	pos.y = positions[3 * id + 1];
	pos.z = positions[3 * id + 2];
	particles_shared[threadIdx.x] = pos;
	__syncthreads();

	unsigned int i;
	for (i = 0; i < BLOCK_SIZE && i < NUM_PARTICLES; i++) { //all particles
		float3 other = particles_shared[i];
		if (curr.x != other.x || curr.y != other.y || curr.z != other.z) { //don't affect own particle
			float3 ray = { curr.x - other.x, curr.y - other.y, curr.z - other.z };
			float dist = ray.x * ray.x + ray.y * ray.y + ray.z * ray.z;
			float xadd = GRAVITY * UNIVERSAL_MASS * (float)ray.x / (dist * dist * dist);
			float yadd = GRAVITY * UNIVERSAL_MASS * (float)ray.y / (dist * dist * dist);
			float zadd = GRAVITY * UNIVERSAL_MASS * (float)ray.z / (dist * dist * dist);
			atomicAdd(&(accel.x), xadd);
			atomicAdd(&(accel.y), yadd);
			atomicAdd(&(accel.z), zadd);
		}
	}
	__syncthreads();
}

__global__
void gravityParallelBaseKernel(float* position, float* velocity, float* acceleration, unsigned int simulationLength) {

	//strategy: one thread per particle

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= NUM_PARTICLES) return;

	//adjust positions and velocities
	int i = 3 * index;

	float3 curPos;
	curPos.x = position[3 * i];
	curPos.y = position[3 * i + 1];
	curPos.z = position[3 * i + 2];
	float3 accel;

	unsigned int simulCount;
	for (simulCount = 0; simulCount < simulationLength; simulCount++) {
		gravityParallelAccelKernel(curPos, position, simulationLength, accel); //accel passed by reference
		cudaCheck(hipDeviceSynchronize());
		__syncthreads(); //all threads (particles) finish calculating acceleration based on physics relative to ALL other particles in system

		position[i] += velocity[i] * EPOCH; //EPOCH is dt
		position[i + 1] += velocity[i + 1] * EPOCH;
		position[i + 2] += velocity[i + 2] * EPOCH;

		velocity[i] += velocity[i] * EPOCH; //EPOCH is dt
		velocity[i + 1] += velocity[i + 1] * EPOCH;
		velocity[i + 2] += velocity[i + 2] * EPOCH;

		velocity[i] += accel.x;
		velocity[i + 1] += accel.y;
		velocity[i + 2] += accel.z;

		__syncthreads(); //all threads (particles) finish current iteration of simulation
	}
	return;
}

void gravityParallel(float* hostPositions, float* hostVelocities, float* hostAccelerations, unsigned int simulationLength) {
	//CUDA prep code
	float* devicePositions;
	float* deviceVelocities;
	float* deviceAccelerations;
	size_t size = NUM_PARTICLES * 3 * sizeof(float);

	cudaCheck(hipSetDevice(0)); //choose GPU
	cudaCheck(hipMalloc((void **)&devicePositions, size));
	cudaCheck(hipMalloc((void **)&deviceVelocities, size));
	cudaCheck(hipMalloc((void **)&deviceAccelerations, size));
	cudaCheck(hipMemcpy(devicePositions, hostPositions, size, hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(deviceVelocities, hostVelocities, size, hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(deviceAccelerations, hostAccelerations, size, hipMemcpyHostToDevice));
	dim3 dimGrid, dimBlock;
	dimGrid.x = (size - 1) / BLOCK_SIZE + 1;
	dimBlock.x = BLOCK_SIZE;
	gravityParallelBaseKernel <<<dimGrid, dimBlock >>>(devicePositions, deviceVelocities, deviceAccelerations, simulationLength);
	cudaCheck(hipDeviceSynchronize());
	cudaCheck(hipMemcpy(hostPositions, devicePositions, size, hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(hostVelocities, deviceVelocities, size, hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(hostAccelerations, deviceAccelerations, size, hipMemcpyDeviceToHost));
	cudaCheck(hipFree(devicePositions));
	cudaCheck(hipFree(deviceVelocities));
	cudaCheck(hipFree(deviceAccelerations));
	
	return;
}

//print particles after a single round of serial and parallel to compare output and check correctness
void particleSystem::gravityBoth(float* positions, float* velocities, float* accelerations, unsigned int numRounds) {
	unsigned int round;
	for (round = 0; round < numRounds; round++) {
		
		//SERIAL PORTION
		this->gravitySerial(1); //execution phase
		this->printParticles(); //print phase

		//PARALLEL PORTION
		gravityParallel(positions, velocities, accelerations, 1); //execution phase
		printParticlcesArrays(positions, velocities, accelerations); //print phase
	}

	//CUDA cleanup code
}

int main()
{

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaCheck(hipDeviceReset());

	particleSystem parSys(NUM_PARTICLES);
	//parSys.printParticles();
	//parSys.gravitySerial(SIMULATION_LENGTH);
	float* pos = parSys.particlesPosfloatArray();
	float* vel = parSys.particlesVelfloatArray();
	float* acc = parSys.particlesAccfloatArray();
	std::cout << std::endl;
	//parSys.printPosfloatArray(pos);
	//parSys.printVelfloatArray(vel);
	parSys.gravityBoth(pos, vel, acc, SIMULATION_LENGTH);

	system("pause"); //see output of terminal
	return 0;
}
